#include "hip/hip_runtime.h"
/*********************************************************************************/
//
// Polybench kernels implementation on CUDA GPU
//
// Computer & Information Science, University of Delaware
// Author(s):   Sudhee Ayalasomayajula (sudhee1@gmail.com)
//              John Cavazos (cavazos@cis.udel.edu)
//		Scott Grauer Gray(sgrauerg@gmail.com)
//              Robert Searles (rsearles35@aol.com)   
//              Lifan Xu (xulifan@udel.edu)
//
// Contact(s): Lifan Xu (xulifan@udel.edu)
// Reference(s):
//
/*********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>

/* Default problem size. */
# define NI 512
# define NJ 512
# define NK 512
# define NL 512

typedef double real;
#define GPU_DEVICE 0

real C[NI][NJ];
real A[NI][NK];
real B[NK][NJ];
real D[NJ][NL];
real E[NI][NL];
real F[NI][NL];

real *A_gpu;
real *B_gpu;
real *C_gpu;
real *D_gpu;
real *E_gpu;

double t_start, t_end;

void init_array()
{
  int i, j;


  for (i = 0; i < NI; i++)
    for (j = 0; j < NK; j++)
      A[i][j] = 1;
  for (i = 0; i < NK; i++)
    for (j = 0; j < NJ; j++)
      B[i][j] = 2;
  for (i = 0; i < NI; i++)
    for (j = 0; j < NJ; j++)
      C[i][j] = 3;
  for (i = 0; i < NJ; i++)
    for (j = 0; j < NL; j++)
      D[i][j] = 4;
  for (i = 0; i < NI; i++)
    for (j = 0; j < NL; j++)
      E[i][j] = 5;
  for (i = 0; i < NI; i++)
    for (j = 0; j < NL; j++)
      F[i][j] = 5;
	
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}



void compareResults(){
	int i,j,fail;
	fail = 0;

	for (i=0; i<NI; i++)
	{
		for (j=0; j<NL; j++)
		{
			if (abs(E[i][j]) < abs(F[i][j])){
				if ((abs(F[i][j]) - abs(E[i][j])) > 0.1){
					fail++;
				}
			}
			else {
				if ((abs(E[i][j]) - abs(F[i][j])) > 0.1){
					fail++;
				}
			}
		}
	}
	
	// print results
	printf("Number of misses: %d\n", fail);
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
	
	return;
}

void GPU_mem_init()
{
	hipMalloc((void **)&A_gpu, sizeof(real) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(real) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(real) * NI * NJ);
	hipMalloc((void **)&D_gpu, sizeof(real) * NJ * NL);
	hipMalloc((void **)&E_gpu, sizeof(real) * NI * NL);
	hipMemcpy(A_gpu, A, sizeof(real) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(real) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(real) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(real) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(real) * NI * NL, hipMemcpyHostToDevice);	
	
}


__global__ void mm2_kernel(real *a, real *b, real *c, int dim0, int dim1, int dim2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int tid = i * dim0 + j;
	for(k=0; k< dim0; k++)
	{
		c[tid] += a[i * dim0 + k] * b[k * dim0 +j];
	}
	
	return;
}

void mm2_cpu(real A[NI][NK], real B[NK][NJ], real C[NI][NJ], real D[NJ][NL], real E[NI][NL], int ni, int nj, int nk, int nl)
{
	int i,j,k;
	
  	for (i = 0; i < ni; i++){
    		for (j = 0; j < nj; j++)
	      {
	
		for (k = 0; k < nk; ++k)
		  C[i][j] += A[i][k] * B[k][j];
	      }
	}
	for (i = 0; i < ni; i++){
		for (j = 0; j < nl; j++)
	      {
	
		for (k = 0; k < nj; ++k)
		  E[i][j] += C[i][k] * D[k][j];
	      }
	}
}

int main(int argc, char** argv)
{

	int ni = NI;
	int nj = NJ;
	int nk = NK;
	int nl = NL;

  	init_array();
	 
	GPU_argv_init();

	GPU_mem_init();
		
	dim3 block(32,8);
	dim3 grid1(NI/32,NJ/8);
	dim3 grid2(NI/32,NL/8);
	t_start = rtclock();
	mm2_kernel<<<grid1,block>>>(A_gpu,B_gpu,C_gpu,ni,nk,nj);
	mm2_kernel<<<grid2,block>>>(C_gpu,D_gpu,E_gpu,ni,nj,nl);
	hipDeviceSynchronize();
	t_end = rtclock();
	hipMemcpy(E, E_gpu, sizeof(real) * NI * NL, hipMemcpyDeviceToHost);

	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	t_start = rtclock();

	mm2_cpu(A, B, C, D, F, ni, nj, nk, nl);

	t_end = rtclock();

	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults();

  return 0;
}
